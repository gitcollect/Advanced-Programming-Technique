#include "hip/hip_runtime.h"
/* 
 * File:   MBSet.cu
 * 
 * Created on June 24, 2012
 * 
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 * 
 */

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"

#include <GL/freeglut.h>

#include <vector>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "matrix.h"
#include "complex.h"


// Size of window in pixels, both width and height
#define WINDOW_DIM            512

using namespace std;
int width=512, height=512, SCALE=1, xt=0,yt=0;
std::vector<Vec2D> points;
std::vector< std::vector<double> > colorss;


// Initial screen coordinates, both host and device.
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);
Complex* dev_minC;
Complex* dev_maxC;
const int maxIt = 2000; // Msximum Iterations
double updateRate =20;

// Define the RGB Class
class RGB
{
public:
  RGB()
    : r(0), g(0), b(0) {}
  RGB(double r0, double g0, double b0)
    : r(r0), g(g0), b(b0) {}
public:
  double r;
  double g;
  double b;
};

RGB* colors = 0; // Array of color values

void InitializeColors()
{
  colors = new RGB[maxIt + 1];
  for (int i = 0; i < maxIt; ++i)
    {
      if (i < 5)
        { // Try this.. just white for small it counts
          colors[i] = RGB(1, 1, 1);
        }
      else
        {
          colors[i] = RGB(drand48(), drand48(), drand48());
        }
    }
  colors[maxIt] = RGB(); // black
}

void computePoints(unsigned int itt)
{
	colorss.clear();
    points.clear();
    ComplexNumber z(0,0), c(0,0);
	unsigned int ittcount;

    for ( double x = -2; x <= 1; x+=0.0059 )
	{
		for ( double y = -1.2; y <= 1.8; y+= 0.0059 )
		{
			c.real=x;c.img=y;
			z.real=0;z.img=0;
			ittcount=0;
			while ( ittcount < itt && z.length() < 4 )
			{
				z = z.multiply(z).add(c);
				ittcount++;
			}
			points.push_back(Vec2D(x,y));
			std::vector<double> color;
			color.push_back(ittcount/itt);
			color.push_back(sin((double)ittcount));
			color.push_back(abs(cos((double)ittcount)));
			colorss.push_back(color);
		}
	}
}

void setup()
{
	// we gonna do this in ortho
	glViewport(0, 0, width, height);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(-width/2., width/2., -height/2., height/2., -1.,1);
	glMatrixMode(GL_MODELVIEW);
}

void resize(int w, int h)
{
	width=w;
	height=h;
	//setup();
	//computePoints(1000);
	setup();
}
void drawfractal()
{
	glBegin(GL_POINTS);
	for ( int i = 0; i < points.size(); i++ ){
		glColor3f(colorss[i][0], colorss[i][1], colorss[i][2]);
		glVertex2f(points[i].x*SCALE,points[i].y*SCALE);
	}
	glEnd();
}
void display()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glClearColor(1.,1.,1.,1);
	glLoadIdentity();
	glTranslatef(xt,-140+yt,0);
	drawfractal();
	glutSwapBuffers();
}
void mouse(int x, int y)
{
}
void mousefunc(int button, int state, int x, int y)
{
}
void keyboard(unsigned char key, int x, int y)
{
	switch(key)
	{
	case 'w':
		yt+=10;
		break;
	case 's':
		yt-=10;
		break;
	case 'a':
		xt-=10;
		break;
	case 'd':
		xt+=10;
		break;
	case 'z':
	case 'Z':
		SCALE*=2;
		break;
	case 'x':
	case 'X':
		SCALE/=2;
		break;
	}
}
void update(int value)
{
	glutPostRedisplay();
	glutTimerFunc(15, update, 0);
}

int main(int argc, char *argv[])
{
	srand(time(NULL));
	computePoints(2000);
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_DEPTH | GLUT_DOUBLE);
	glutInitWindowSize(width,height);
	glutCreateWindow("MBSet");
	setup();
	glutDisplayFunc(display);
	glutReshapeFunc(resize);
	glutPassiveMotionFunc(mouse);
	glutMouseFunc(mousefunc);
	glutKeyboardFunc(keyboard);
	glutTimerFunc(25, update, 0);
	glEnable(GL_POINT_SMOOTH);
	glHint(GL_POINT_SMOOTH_HINT, GL_NICEST);
	glPointSize(1);
	glutMainLoop();
    return 0;
}
