#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <GL/glut.h>

#include "global.h"

#define blockWidth 8
#define blockHeight 8

//extern double xmin, xmax;
//extern double ymin, ymax;
//extern int XSize, YSize;
//extern const int MAX_ITER_NUM;

//extern ParameterStruct paras;


__global__ void kernel(uchar4 * pbo, uchar4 * palette, ParameterStruct paras)
{
	int indx = (blockIdx.x*blockDim.x) + threadIdx.x;
	int indy = (blockIdx.y*blockDim.y) + threadIdx.y;
	
	if (indx >= paras.xSize || indy >= paras.ySize)
		return;

	int index = indx * paras.xSize + indy;

	pbo[index] = palette[index];
}


//extern const int blockWidth = 8;
//extern const int blockHeight = 8;
//extern const int numBlocksWidth = paras.xSize/blockWidth;
//extern const int numBlocksHeight = paras.ySize/blockHeight;
// Declare the cuda dimension parameters
//extern dim3 blockSize(blockWidth, blockHeight);
//extern dim3 numBlocks(numBlocksWidth, numBlocksHeight);

extern "C" void run_kernel(uchar4 *pbo, uchar4 * palette, ParameterStruct paras)
{
	uchar4 * d_palette;
	hipMalloc(&d_palette, sizeof(uchar4)*(paras.maxIterNum+1)); 
	hipMemcpy(d_palette, palette, sizeof(uchar4)*(paras.maxIterNum+1), hipMemcpyHostToDevice);

	const int numBlocksWidth = paras.xSize/blockWidth;
	const int numBlocksHeight = paras.ySize/blockHeight;
	dim3 blockSize(blockWidth, blockHeight);
	dim3 numBlocks(numBlocksWidth, numBlocksHeight);

	// Call the kernel
	kernel<<<numBlocks,blockSize>>>(pbo, d_palette, paras);
	hipFree(d_palette);
}

extern "C" void runCuda(struct hipGraphicsResource **pbo_resource, uchar4 *palette, ParameterStruct paras)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    hipGraphicsMapResources(1, pbo_resource, 0);
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *pbo_resource);
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

    run_kernel((uchar4*)dptr, palette, paras);

    // unmap buffer object
    hipGraphicsUnmapResources(1, pbo_resource, 0);
}

extern "C" void createPBO(GLuint *pbo, struct hipGraphicsResource **pbo_res, unsigned int pbo_res_flags, ParameterStruct paras)
{
    //assert(pbo);

    // create buffer object
    glGenBuffers(1, pbo);
    glBindBuffer(GL_ARRAY_BUFFER, *pbo);

    // initialize buffer object
    unsigned int size = paras.xSize * paras.ySize * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    hipGraphicsGLRegisterBuffer(pbo_res, *pbo, pbo_res_flags);

    SDK_CHECK_ERROR_GL();
}
